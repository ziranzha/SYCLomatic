// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0
// RUN: dpct --format-range=none -out-root %T/cusolver_helper_function_4 %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cusolver_helper_function_4/cusolver_helper_function_4.dp.cpp --match-full-lines %s

//CHECK:#include <sycl/sycl.hpp>
//CHECK-NEXT:#include <dpct/dpct.hpp>
//CHECK-NEXT:#include <dpct/lapack_utils.hpp>

#include <hip/hip_runtime.h>
#include "hipsolver.h"

int foo1() {
  float* a_s;
  double* a_d;
  float2* a_c;
  double2* a_z;
  float* w_s;
  double* w_d;
  float* w_c;
  double* w_z;

  hipsolverHandle_t handle;
  hipsolverDnParams_t params;

  int lwork_s;
  int lwork_d;
  int lwork_c;
  int lwork_z;

  int h_meig_s;
  int h_meig_d;
  int h_meig_c;
  int h_meig_z;

  //CHECK:dpct::lapack::syheevx_scratchpad_size<float, float>(*handle, oneapi::mkl::job::vec, oneapi::mkl::rangev::all, oneapi::mkl::uplo::upper, 2, 2, 0, 0, 0, 0, &lwork_s);
  //CHECK-NEXT:dpct::lapack::syheevx_scratchpad_size<double, double>(*handle, oneapi::mkl::job::vec, oneapi::mkl::rangev::all, oneapi::mkl::uplo::upper, 2, 2, 0, 0, 0, 0, &lwork_d);
  //CHECK-NEXT:dpct::lapack::syheevx_scratchpad_size<sycl::float2, float>(*handle, oneapi::mkl::job::vec, oneapi::mkl::rangev::all, oneapi::mkl::uplo::upper, 2, 2, 0, 0, 0, 0, &lwork_c);
  //CHECK-NEXT:dpct::lapack::syheevx_scratchpad_size<sycl::double2, double>(*handle, oneapi::mkl::job::vec, oneapi::mkl::rangev::all, oneapi::mkl::uplo::upper, 2, 2, 0, 0, 0, 0, &lwork_z);
  hipsolverDnSsyevdx_bufferSize(handle, HIPSOLVER_EIG_MODE_VECTOR, HIPSOLVER_EIG_RANGE_ALL, HIPBLAS_FILL_MODE_UPPER, 2, a_s, 2, 0, 0, 0, 0, &h_meig_s, w_s, &lwork_s);
  hipsolverDnDsyevdx_bufferSize(handle, HIPSOLVER_EIG_MODE_VECTOR, HIPSOLVER_EIG_RANGE_ALL, HIPBLAS_FILL_MODE_UPPER, 2, a_d, 2, 0, 0, 0, 0, &h_meig_d, w_d, &lwork_d);
  hipsolverDnCheevdx_bufferSize(handle, HIPSOLVER_EIG_MODE_VECTOR, HIPSOLVER_EIG_RANGE_ALL, HIPBLAS_FILL_MODE_UPPER, 2, a_c, 2, 0, 0, 0, 0, &h_meig_c, w_c, &lwork_c);
  hipsolverDnZheevdx_bufferSize(handle, HIPSOLVER_EIG_MODE_VECTOR, HIPSOLVER_EIG_RANGE_ALL, HIPBLAS_FILL_MODE_UPPER, 2, a_z, 2, 0, 0, 0, 0, &h_meig_z, w_z, &lwork_z);

  float* device_ws_s;
  double* device_ws_d;
  float2* device_ws_c;
  double2* device_ws_z;

  int *info;

  //CHECK:dpct::lapack::syheevx<float, float>(*handle, oneapi::mkl::job::vec, oneapi::mkl::rangev::all, oneapi::mkl::uplo::upper, 2, a_s, 2, 0, 0, 0, 0, &h_meig_s, w_s, device_ws_s, lwork_s, info);
  //CHECK-NEXT:dpct::lapack::syheevx<double, double>(*handle, oneapi::mkl::job::vec, oneapi::mkl::rangev::all, oneapi::mkl::uplo::upper, 2, a_d, 2, 0, 0, 0, 0, &h_meig_d, w_d, device_ws_d, lwork_d, info);
  //CHECK-NEXT:dpct::lapack::syheevx<sycl::float2, float>(*handle, oneapi::mkl::job::vec, oneapi::mkl::rangev::all, oneapi::mkl::uplo::upper, 2, a_c, 2, 0, 0, 0, 0, &h_meig_c, w_c, device_ws_c, lwork_c, info);
  //CHECK-NEXT:dpct::lapack::syheevx<sycl::double2, double>(*handle, oneapi::mkl::job::vec, oneapi::mkl::rangev::all, oneapi::mkl::uplo::upper, 2, a_z, 2, 0, 0, 0, 0, &h_meig_z, w_z, device_ws_z, lwork_z, info);
  hipsolverDnSsyevdx(handle, HIPSOLVER_EIG_MODE_VECTOR, HIPSOLVER_EIG_RANGE_ALL, HIPBLAS_FILL_MODE_UPPER, 2, a_s, 2, 0, 0, 0, 0, &h_meig_s, w_s, device_ws_s, lwork_s, info);
  hipsolverDnDsyevdx(handle, HIPSOLVER_EIG_MODE_VECTOR, HIPSOLVER_EIG_RANGE_ALL, HIPBLAS_FILL_MODE_UPPER, 2, a_d, 2, 0, 0, 0, 0, &h_meig_d, w_d, device_ws_d, lwork_d, info);
  hipsolverDnCheevdx(handle, HIPSOLVER_EIG_MODE_VECTOR, HIPSOLVER_EIG_RANGE_ALL, HIPBLAS_FILL_MODE_UPPER, 2, a_c, 2, 0, 0, 0, 0, &h_meig_c, w_c, device_ws_c, lwork_c, info);
  hipsolverDnZheevdx(handle, HIPSOLVER_EIG_MODE_VECTOR, HIPSOLVER_EIG_RANGE_ALL, HIPBLAS_FILL_MODE_UPPER, 2, a_z, 2, 0, 0, 0, 0, &h_meig_z, w_z, device_ws_z, lwork_z, info);

  return 0;
}

int foo2() {
  float* a_s;
  double* a_d;
  float2* a_c;
  double2* a_z;
  float* b_s;
  double* b_d;
  float2* b_c;
  double2* b_z;
  float* w_s;
  double* w_d;
  float* w_c;
  double* w_z;

  hipsolverHandle_t handle;
  hipsolverDnParams_t params;

  int lwork_s;
  int lwork_d;
  int lwork_c;
  int lwork_z;

  int h_meig_s;
  int h_meig_d;
  int h_meig_c;
  int h_meig_z;

  //CHECK:dpct::lapack::syhegvx_scratchpad_size<float, float>(*handle, 1, oneapi::mkl::job::vec, oneapi::mkl::rangev::all, oneapi::mkl::uplo::upper, 2, 2, 2, 0, 0, 0, 0, &lwork_s);
  //CHECK-NEXT:dpct::lapack::syhegvx_scratchpad_size<double, double>(*handle, 1, oneapi::mkl::job::vec, oneapi::mkl::rangev::all, oneapi::mkl::uplo::upper, 2, 2, 2, 0, 0, 0, 0, &lwork_d);
  //CHECK-NEXT:dpct::lapack::syhegvx_scratchpad_size<sycl::float2, float>(*handle, 1, oneapi::mkl::job::vec, oneapi::mkl::rangev::all, oneapi::mkl::uplo::upper, 2, 2, 2, 0, 0, 0, 0, &lwork_c);
  //CHECK-NEXT:dpct::lapack::syhegvx_scratchpad_size<sycl::double2, double>(*handle, 1, oneapi::mkl::job::vec, oneapi::mkl::rangev::all, oneapi::mkl::uplo::upper, 2, 2, 2, 0, 0, 0, 0, &lwork_z);
  hipsolverDnSsygvdx_bufferSize(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPSOLVER_EIG_RANGE_ALL, HIPBLAS_FILL_MODE_UPPER, 2, a_s, 2, b_s, 2, 0, 0, 0, 0, &h_meig_s, w_s, &lwork_s);
  hipsolverDnDsygvdx_bufferSize(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPSOLVER_EIG_RANGE_ALL, HIPBLAS_FILL_MODE_UPPER, 2, a_d, 2, b_d, 2, 0, 0, 0, 0, &h_meig_d, w_d, &lwork_d);
  hipsolverDnChegvdx_bufferSize(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPSOLVER_EIG_RANGE_ALL, HIPBLAS_FILL_MODE_UPPER, 2, a_c, 2, b_c, 2, 0, 0, 0, 0, &h_meig_c, w_c, &lwork_c);
  hipsolverDnZhegvdx_bufferSize(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPSOLVER_EIG_RANGE_ALL, HIPBLAS_FILL_MODE_UPPER, 2, a_z, 2, b_z, 2, 0, 0, 0, 0, &h_meig_z, w_z, &lwork_z);

  float* device_ws_s;
  double* device_ws_d;
  float2* device_ws_c;
  double2* device_ws_z;

  int *info;

  //CHECK:dpct::lapack::syhegvx<float, float>(*handle, 1, oneapi::mkl::job::vec, oneapi::mkl::rangev::all, oneapi::mkl::uplo::upper, 2, a_s, 2, b_s, 2, 0, 0, 0, 0, &h_meig_s, w_s, device_ws_s, lwork_s, info);
  //CHECK-NEXT:dpct::lapack::syhegvx<double, double>(*handle, 1, oneapi::mkl::job::vec, oneapi::mkl::rangev::all, oneapi::mkl::uplo::upper, 2, a_d, 2, b_d, 2, 0, 0, 0, 0, &h_meig_d, w_d, device_ws_d, lwork_d, info);
  //CHECK-NEXT:dpct::lapack::syhegvx<sycl::float2, float>(*handle, 1, oneapi::mkl::job::vec, oneapi::mkl::rangev::all, oneapi::mkl::uplo::upper, 2, a_c, 2, b_c, 2, 0, 0, 0, 0, &h_meig_c, w_c, device_ws_c, lwork_c, info);
  //CHECK-NEXT:dpct::lapack::syhegvx<sycl::double2, double>(*handle, 1, oneapi::mkl::job::vec, oneapi::mkl::rangev::all, oneapi::mkl::uplo::upper, 2, a_z, 2, b_z, 2, 0, 0, 0, 0, &h_meig_z, w_z, device_ws_z, lwork_z, info);
  hipsolverDnSsygvdx(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPSOLVER_EIG_RANGE_ALL, HIPBLAS_FILL_MODE_UPPER, 2, a_s, 2, b_s, 2, 0, 0, 0, 0, &h_meig_s, w_s, device_ws_s, lwork_s, info);
  hipsolverDnDsygvdx(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPSOLVER_EIG_RANGE_ALL, HIPBLAS_FILL_MODE_UPPER, 2, a_d, 2, b_d, 2, 0, 0, 0, 0, &h_meig_d, w_d, device_ws_d, lwork_d, info);
  hipsolverDnChegvdx(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPSOLVER_EIG_RANGE_ALL, HIPBLAS_FILL_MODE_UPPER, 2, a_c, 2, b_c, 2, 0, 0, 0, 0, &h_meig_c, w_c, device_ws_c, lwork_c, info);
  hipsolverDnZhegvdx(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPSOLVER_EIG_RANGE_ALL, HIPBLAS_FILL_MODE_UPPER, 2, a_z, 2, b_z, 2, 0, 0, 0, 0, &h_meig_z, w_z, device_ws_z, lwork_z, info);

  return 0;
}

int foo3() {
  float* a_s;
  double* a_d;
  float2* a_c;
  double2* a_z;
  float* b_s;
  double* b_d;
  float2* b_c;
  double2* b_z;
  float* w_s;
  double* w_d;
  float* w_c;
  double* w_z;

  hipsolverHandle_t handle;
  hipsolverSyevjInfo_t params;

  int lwork_s;
  int lwork_d;
  int lwork_c;
  int lwork_z;

  //CHECK:dpct::lapack::syhegvd_scratchpad_size<float>(*handle, 1, oneapi::mkl::job::vec, oneapi::mkl::uplo::upper, 2, 2, 2, &lwork_s);
  //CHECK-NEXT:dpct::lapack::syhegvd_scratchpad_size<float>(*handle, 1, oneapi::mkl::job::vec, oneapi::mkl::uplo::upper, 2, 2, 2, &lwork_d);
  //CHECK-NEXT:dpct::lapack::syhegvd_scratchpad_size<sycl::float2>(*handle, 1, oneapi::mkl::job::vec, oneapi::mkl::uplo::upper, 2, 2, 2, &lwork_c);
  //CHECK-NEXT:dpct::lapack::syhegvd_scratchpad_size<sycl::double2>(*handle, 1, oneapi::mkl::job::vec, oneapi::mkl::uplo::upper, 2, 2, 2, &lwork_z);
  hipsolverDnSsygvj_bufferSize(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 2, a_s, 2, b_s, 2, w_s, &lwork_s, params);
  hipsolverDnDsygvj_bufferSize(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 2, a_d, 2, b_d, 2, w_d, &lwork_d, params);
  hipsolverDnChegvj_bufferSize(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 2, a_c, 2, b_c, 2, w_c, &lwork_c, params);
  hipsolverDnZhegvj_bufferSize(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 2, a_z, 2, b_z, 2, w_z, &lwork_z, params);

  float* device_ws_s;
  double* device_ws_d;
  float2* device_ws_c;
  double2* device_ws_z;

  int *info;

  //CHECK:dpct::lapack::syhegvd<float, float>(*handle, 1, oneapi::mkl::job::vec, oneapi::mkl::uplo::upper, 2, a_s, 2, b_s, 2, w_s, device_ws_s, lwork_s, info);
  //CHECK-NEXT:dpct::lapack::syhegvd<double, double>(*handle, 1, oneapi::mkl::job::vec, oneapi::mkl::uplo::upper, 2, a_d, 2, b_d, 2, w_d, device_ws_d, lwork_d, info);
  //CHECK-NEXT:dpct::lapack::syhegvd<sycl::float2, float>(*handle, 1, oneapi::mkl::job::vec, oneapi::mkl::uplo::upper, 2, a_c, 2, b_c, 2, w_c, device_ws_c, lwork_c, info);
  //CHECK-NEXT:dpct::lapack::syhegvd<sycl::double2, double>(*handle, 1, oneapi::mkl::job::vec, oneapi::mkl::uplo::upper, 2, a_z, 2, b_z, 2, w_z, device_ws_z, lwork_z, info);
  hipsolverDnSsygvj(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 2, a_s, 2, b_s, 2, w_s, device_ws_s, lwork_s, info, params);
  hipsolverDnDsygvj(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 2, a_d, 2, b_d, 2, w_d, device_ws_d, lwork_d, info, params);
  hipsolverDnChegvj(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 2, a_c, 2, b_c, 2, w_c, device_ws_c, lwork_c, info, params);
  hipsolverDnZhegvj(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 2, a_z, 2, b_z, 2, w_z, device_ws_z, lwork_z, info, params);

  return 0;
}
